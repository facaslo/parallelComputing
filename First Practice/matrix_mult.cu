#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <omp.h>
#include <stdbool.h>
#include <math.h>

#define MATRIX_SIZE 1024
#define BLOCK_SIZE 8
#define THREADS 8
#define MAX_DOUBLE 1.7976931348623158E+3


double RandomReal(double low, double high)
{
  double d;
  d = (double) rand() / ((double) RAND_MAX + 1);
  return (low + d * (high - low));
}

void fill_matrix(double *matrix, int n){
    for (int i = 0; i < n * n; i++) {
        *(matrix + i) = RandomReal(-MAX_DOUBLE, MAX_DOUBLE) ;
    }
}

void print_matrix(double *matrix, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%.3f ", *(matrix + i * n + j));
        }
        printf("\n");
    }
}

void multiply_matrices(double *matrix1, double *matrix2, double *result, int n) {
    for (int i = 0; i < n; i++) {
        int row = i * n;
        for (int j = 0; j < n; j++) {
            double sum = 0;
            for (int k = 0; k < n; k++) {
                sum += *(matrix1 + row + k) * *(matrix2 + k * n + j);
            }
            *(result + row + j) = sum;
        }
    }
}

__global__ void matrixMul(double *a, double *b, double *c, int size)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < size && col < size) {
        double sum = 0.0;
        for (int k = 0; k < size; k++) {
            sum += *(a+ row*size + k) * *(b + size*k + col);            
        }
        *(c + size*row + col) = sum;
    }
    //printf("Block id x : %d , Block id y : %d",blockIdx.x, blockIdx.y);    
}

bool compare_matrices(double *matrix1, double *matrix2, int n){
  bool isTheSame = true;
  for (int i=0; i< n; i++){
    int row = i*n;
    for(int j=0; j<n; j++){
      double difference = *(matrix1 + row + j) - *(matrix2 + row + j);
      if(abs(difference) > 1E-6){
        isTheSame = false;
        break;
      }
    }
  }
  return isTheSame;
}


int main()
{
    double *a, *b, *c, *d;
    double *dev_a, *dev_b, *dev_c;
    int matrix_bytes = MATRIX_SIZE * MATRIX_SIZE * sizeof(double);

    // Allocate host memory
    a = (double*)malloc(matrix_bytes);
    b = (double*)malloc(matrix_bytes);
    c = (double*)malloc(matrix_bytes);
    d = (double*)malloc(matrix_bytes);
    // Initialize matrices with random doubles
    srand(time(NULL));
    fill_matrix(a,MATRIX_SIZE);
    fill_matrix(b,MATRIX_SIZE);

    // Allocate device memory
    hipMalloc((void**)&dev_a, matrix_bytes);
    hipMalloc((void**)&dev_b, matrix_bytes);
    hipMalloc((void**)&dev_c, matrix_bytes);

    // Copy matrices to device
    hipMemcpy(dev_a, a, matrix_bytes, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, matrix_bytes, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 gridDim(ceil((float)MATRIX_SIZE / BLOCK_SIZE), ceil((float)MATRIX_SIZE / BLOCK_SIZE), 1);
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, 1);

    // Launch kernel
    matrixMul<<<gridDim, blockDim>>>(dev_a, dev_b, dev_c, MATRIX_SIZE);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
      printf("Error: %s\n", hipGetErrorString(err));
    hipDeviceSynchronize();
    // Copy result back to host
    hipMemcpy(c, dev_c, matrix_bytes, hipMemcpyDeviceToHost);

    // Sequential result
    
    multiply_matrices(a,b,d,MATRIX_SIZE);
    // print_matrix(a,MATRIX_SIZE);
    // print_matrix(b,MATRIX_SIZE);
    // printf("-------------------------------------------------------------------------------\n");
    // print_matrix(c,MATRIX_SIZE);
    // printf("-------------------------------------------------------------------------------\n");
    // print_matrix(d,MATRIX_SIZE);

    bool comparison_result = compare_matrices(c,d,MATRIX_SIZE);
    
    // Free memory
    free(a);
    free(b);
    free(c);
    free(d);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    printf("Matrices iguales: %s \n", comparison_result ? "true" : "false");
    return 0;
}
