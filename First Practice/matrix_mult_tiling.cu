#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <omp.h>
#include <stdbool.h>
#include <math.h>

#define MAX_DOUBLE 1.7976931348623158E+3
#define TILE_WIDTH 2

double RandomReal(double low, double high)
{
  double d;
  d = (double) rand() / ((double) RAND_MAX + 1);
  return (low + d * (high - low));
}

void fill_matrix(double *matrix, int n){
    for (int i = 0; i < n * n; i++) {
        *(matrix + i) = RandomReal(-MAX_DOUBLE, MAX_DOUBLE) ;
    }
}

void print_matrix(double *matrix, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%.3f ", *(matrix + i * n + j));
        }
        printf("\n");
    }
}

void multiply_matrices(double *matrix1, double *matrix2, double *result, int n) {
    for (int i = 0; i < n; i++) {
        int row = i * n;
        for (int j = 0; j < n; j++) {
            double sum = 0;
            for (int k = 0; k < n; k++) {
                sum += *(matrix1 + row + k) * *(matrix2 + k * n + j);
            }
            *(result + row + j) = sum;
        }
    }
}

__global__ void MatrixMulKernel(double* d_M, double* d_N, double* d_P, int Width) {
  
  __shared__ double Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ double Nds[TILE_WIDTH][TILE_WIDTH];
  int bx = blockIdx.x; int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
  // Identify the row and column of the d_P element to work on
  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;
  double Pvalue = 0;
  // Loop over the d_M and d_N tiles required to compute d_P element
  for (int m = 0; m < Width/TILE_WIDTH; ++m) {
    // Coolaborative loading of d_M and d_N tiles into shared memory
    Mds[ty][tx] = *(d_M + Row*Width + m*TILE_WIDTH + tx);
    Nds[ty][tx] = *(d_N + (m*TILE_WIDTH + ty)*Width + Col);
    // Mds[ty][tx] = d_M[Row*Width + m*TILE_WIDTH + tx];
    // Nds[ty][tx] = d_N[(m*TILE_WIDTH + ty)*Width + Col];
    __syncthreads();
    for (int k = 0; k < TILE_WIDTH; ++k) {
      Pvalue += Mds[ty][k] * Nds[k][tx];
    }
    __syncthreads();
  }
  //d_P[Row*Width + Col] = Pvalue;
  *(d_P + Row*Width + Col) = Pvalue;
}

bool compare_matrices(double *matrix1, double *matrix2, int n){
  bool isTheSame = true;
  for (int i=0; i< n; i++){
    int row = i*n;
    for(int j=0; j<n; j++){
      double difference = *(matrix1 + row + j) - *(matrix2 + row + j);
      if(abs(difference) > 1E-3){
        isTheSame = false;
        printf("row:%d - col:%d - difference:%.3f \n", row,col,difference);
        return isTheSame;
      }
    }
  }
  return isTheSame;
}

int main(int argc, char *argv[])
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int matrix_size = atoi(argv[1]);
    int tile_width = atoi(argv[2]);        
    double *a, *b, *c, *d;
    double *dev_a, *dev_b, *dev_c;
    int matrix_bytes = matrix_size * matrix_size * sizeof(double);

    // Allocate host memory
    a = (double*)malloc(matrix_bytes);
    b = (double*)malloc(matrix_bytes);
    c = (double*)malloc(matrix_bytes);
    d = (double*)malloc(matrix_bytes);
    // Initialize matrices with random doubles
    srand(time(NULL));
    fill_matrix(a,matrix_size);
    fill_matrix(b,matrix_size);

    // Allocate device memory
    hipMalloc((void**)&dev_a, matrix_bytes);
    hipMalloc((void**)&dev_b, matrix_bytes);
    hipMalloc((void**)&dev_c, matrix_bytes);

    // Copy matrices to device
    hipMemcpy(dev_a, a, matrix_bytes, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, matrix_bytes, hipMemcpyHostToDevice);

    // Launch kernel
    dim3 gridDim(ceil((float)matrix_size/tile_width), ceil((float)matrix_size/tile_width) ,1);
    dim3 blockDim(tile_width,tile_width,1);
    hipEventRecord(start);
    MatrixMulKernel<<<gridDim, blockDim>>>(dev_a, dev_b, dev_c, matrix_size);
    hipEventRecord(stop);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
      printf("Error: %s\n", hipGetErrorString(err));
    hipDeviceSynchronize();
    // Copy result back to host
    hipMemcpy(c, dev_c, matrix_bytes, hipMemcpyDeviceToHost);

    // Sequential result
    
    multiply_matrices(a,b,d,matrix_size);
    // print_matrix(a,matrix_size);
    // print_matrix(b,matrix_size);
    // printf("-------------------------------------------------------------------------------\n");
    // print_matrix(c,matrix_size);
    // printf("-------------------------------------------------------------------------------\n");
    // print_matrix(d,matrix_size);

    bool comparison_result = compare_matrices(c,d,matrix_size);
    printf("Are matrices the same: %s \n", comparison_result==true?"verdadero":"falso");
    
    // Free memory
    free(a);
    free(b);
    free(c);
    free(d);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    
    int number_of_blocks = ceil((float)matrix_size/tile_width)*ceil((float)matrix_size/tile_width);
    
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Matrix-size:%d - threads per block :%d - Number of blocks:%d - Time:%.20f mS", matrix_size , tile_width* tile_width , number_of_blocks ,  milliseconds);
    return 0;
}
